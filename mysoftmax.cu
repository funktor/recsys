#include <hip/hip_runtime.h>

#include <stdio.h>
#include <unistd.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <array>
#include <map>
#include <set>
#include <unordered_map>
#include <unordered_set>
#include <deque>
#include <tuple>
#include <map>
#include <fcntl.h>
#include <functional>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <string>
#include <random>
#include <algorithm>
#include <chrono>
#include <mutex>
#include <thread>
#include <ctime> 
#include <stdbool.h>    // bool type
#include <fstream>
#include <cmath>
#include <variant>
#include <omp.h>
#include <math.h>
#include <assert.h>

#define BLOCK_WIDTH_PER_DIM 32

void generate_data(float *x, unsigned int n, unsigned int m) {
    std::random_device rd;
    std::mt19937 engine(rd());

    std::uniform_real_distribution<float> dist(0.0, 1.0);

    for (unsigned int i = 0; i < n; i++) {
        for (unsigned int j = 0; j < m; j++) x[i*m+j] = dist(engine);
    }
}

void print_vector(float *x, int start, int end) {
    std::cout << "[";
    for (int i = start; i <= end; i++) std::cout << x[i] << ", ";
    std::cout << "]" << std::endl;
}

__device__ __forceinline__ float atomicMaxF32(float *address, float val) {
    int ret = __float_as_int(*address);
    while(val > __int_as_float(ret))
    {
        int old = ret;
        if((ret = atomicCAS((int *)address, old, __float_as_int(val))) == old)
            break;
    }
    return __int_as_float(ret);
}

__global__
void softmax_cuda(float *inp, float *out, const unsigned long n, const unsigned long m) {
    unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned long p = (m+BLOCK_WIDTH_PER_DIM-1)/BLOCK_WIDTH_PER_DIM;
    extern __shared__ float inp_shared[];

    if (row < n) {
        if (threadIdx.x == 0) {
            inp_shared[threadIdx.y] = -MAXFLOAT;
            inp_shared[BLOCK_WIDTH_PER_DIM + threadIdx.y] = 0.0f;
        }
        __syncthreads();

        for (unsigned long j = threadIdx.x; j < p*BLOCK_WIDTH_PER_DIM; j += p) {
            if (j < m) {
                atomicMaxF32(&inp_shared[threadIdx.y], inp[row*m + j]);
            }
        }

        __syncthreads();

        for (unsigned long j = threadIdx.x; j < p*BLOCK_WIDTH_PER_DIM; j += p) {
            if (j < m) {
                atomicAdd(&inp_shared[BLOCK_WIDTH_PER_DIM + threadIdx.y], exp(inp[row*m + j]-inp_shared[threadIdx.y]));
            }
        }

        __syncthreads();

        for (unsigned long j = threadIdx.x; j < p*BLOCK_WIDTH_PER_DIM; j += p) {
            if (j < m) {
                out[row*m + j] = exp(inp[row*m + j]-inp_shared[threadIdx.y])/inp_shared[BLOCK_WIDTH_PER_DIM + threadIdx.y];
            }
        }
    }
}

void softmax_cuda_launcher(float *inp, float *out, const unsigned long n, const unsigned long m) {
    dim3 bd(BLOCK_WIDTH_PER_DIM, BLOCK_WIDTH_PER_DIM, 1);
    dim3 gd(1, (n+BLOCK_WIDTH_PER_DIM-1)/BLOCK_WIDTH_PER_DIM, 1);

    softmax_cuda<<<gd, bd, 2*BLOCK_WIDTH_PER_DIM*sizeof(float)>>>(inp, out, n, m);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
}

int main(int argc, char *argv[]) {
    unsigned int n = 10000;
    unsigned int m = 1000;

    float *x, *y;

    hipMallocManaged(&x, n*m*sizeof(float));
    hipMallocManaged(&y, n*m*sizeof(float));

    generate_data(x, n, m);

    auto start = std::chrono::high_resolution_clock::now();
    softmax_cuda_launcher(x, y, n, m);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);

    std::cout << "Duration = " << duration.count() << " ms" << std::endl;
}